
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

/*
Multiply two matrices
*/
__global__ void multiplyMatrices(float *matrix_a, float *matrix_b, float *matrix_c, int rows_a,
                                 int cols_a, int cols_b) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows_a && col < cols_b) {
        float sum = 0.0f;
        for (int i = 0; i < cols_a; i++) {
            sum += matrix_a[row * cols_a + i] * matrix_b[i * cols_b + col];
        }
        matrix_c[row * cols_b + col] = sum;
    }
}

float *allocateGPUMatrix(int rows, int cols) {
    float *matrix;
    hipError_t err = hipMallocManaged(&matrix, rows * cols * sizeof(float));
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        return NULL;
    }
    return matrix;
}

float *allocateMatrix(int rows, int cols) { return (float *)malloc(rows * cols * sizeof(float)); }

void setMatrixIdentity(float *matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            matrix[i * cols + j] = (i == j) ? 1.0f : 0.0f;
        }
    }
}

void setMatrixRandom(float *matrix, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        matrix[i] = (float)rand() / RAND_MAX;
    }
}

void printMatrix(float *matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%*.1f ", 4, matrix[i * cols + j]);
        }
        puts("");
    }
}

void cpuMultiplyMatrices(float *matrix_a, float *matrix_b, float *matrix_c, int rows_a, int cols_a,
                         int cols_b) {
    for (int row = 0; row < rows_a; ++row) {
        for (int col = 0; col < cols_b; ++col) {
            float sum = 0.0f;
            for (int i = 0; i < cols_a; ++i) {
                sum += matrix_a[row * cols_a + i] * matrix_b[i * cols_b + col];
            }
            matrix_c[row * cols_b + col] = sum;
        }
    }
}

void testGPUMultiplication(int n) {
    float *matrix_a = allocateGPUMatrix(n, n);
    float *matrix_b = allocateGPUMatrix(n, n);
    float *matrix_c = allocateGPUMatrix(n, n);

    // Interestingly, there's no performance difference between multiplying
    // identity matrices and random matrices
    setMatrixRandom(matrix_a, n, n);
    setMatrixRandom(matrix_b, n, n);

    hipEvent_t multiplication_start, multiplication_end;
    hipEventCreate(&multiplication_start);
    hipEventCreate(&multiplication_end);

    hipEventRecord(multiplication_start);

    const int block_size = 32;
    const dim3 grid_dim((n + block_size - 1) / block_size, (n + block_size - 1) / block_size);
    const dim3 block_dim(block_size, block_size);

    // Takes just 3ms for n=1000
    multiplyMatrices<<<grid_dim, block_dim>>>(matrix_a, matrix_b, matrix_c, n, n, n);

    hipEventRecord(multiplication_end);
    hipDeviceSynchronize();

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, multiplication_start, multiplication_end);
    printf("GPU multiplied in %.3f milliseconds\n", milliseconds);

    if (n < 10) {
        printMatrix(matrix_c, n, n);
    }

    // Let's be nice and clean up
    hipEventDestroy(multiplication_start);
    hipEventDestroy(multiplication_end);

    hipFree(matrix_a);
    hipFree(matrix_b);
    hipFree(matrix_c);
}

void testCPUMultiplication(int n) {
    float *matrix_a = allocateMatrix(n, n);
    float *matrix_b = allocateMatrix(n, n);
    float *matrix_c = allocateMatrix(n, n);

    setMatrixRandom(matrix_a, n, n);
    setMatrixRandom(matrix_b, n, n);

    struct timeval start, end;

    gettimeofday(&start, NULL);
    // Takes 553ms for n=1000 with -O3 optimizations
    cpuMultiplyMatrices(matrix_a, matrix_b, matrix_c, n, n, n);
    gettimeofday(&end, NULL);

    double elapsed_milliseconds =
        (end.tv_sec - start.tv_sec) * 1000.0 + (end.tv_usec - start.tv_usec) / 1000.0;
    printf("CPU multiplied in %.3f milliseconds\n", elapsed_milliseconds);

    free(matrix_a);
    free(matrix_b);
    free(matrix_c);
}

int main() {
    int n = 1000;

    testGPUMultiplication(n);
    testCPUMultiplication(n);

    return 0;
}
